#include "hip/hip_runtime.h"
#include "sample.h"

template <typename scalar_t>
__global__ void fK() {};

template <typename scalar_t>
__global__ void bK() {};

template <typename scalar_t, unsigned int B, unsigned int T, hipStream_t S>
torch::Tensor forward() {
    // CHECK_INPUT(sample);
    // torch::Tensor output = torch::empty_like()

    AT_DISPATCH_FLOATING_TYPES(DtypeTraits<scalar_t>.torch_t, "forward", ([&] {
        fK<scalar_t><<<B, T, 0, S>>>();
    }));
    CUDA_CHECK_KERNEL();

    return output;
}

template <typename scalar_t, unsigned int B, unsigned int T, hipStream_t S>
torch::Tensor backward() {
    //CHECK_INPUT(sample);
    // torch::Tensor grad_input = torch::empty_like()

    AT_DISPATCH_FLOATING_TYPES(DtypeTraits<scalar_t>.torch_t, "backward", ([&] {
        bK<scalar_t><<<B, T, 0, S>>>();
    }));
    CUDA_CHECK_KERNEL();

    return grad_input;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "forward pass");
    m.def("backward", &backward, "backward pass");
}
